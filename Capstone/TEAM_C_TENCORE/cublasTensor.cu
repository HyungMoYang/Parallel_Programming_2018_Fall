/*
cuBLAS users will notice a few changes from their existing cuBLAS GEMM code:

1.The routine must be a GEMM; currently, only GEMMs support Tensor Core execution.

2.The math mode must be set to HIPBLAS_TENSOR_OP_MATH. Floating point math is not associative, so 
the results of the Tensor Core math routines are not quite bit-equivalent to the results of the analogous 
non-Tensor Core math routines.  cuBLAS requires the user to “opt in” to the use of tensor cores.
3.All of k, lda, ldb, and ldc must be a multiple of eight; 
m must be a multiple of four. The Tensor Core math routines stride through input data in steps of
eight values, so the dimensions of the matrices must be multiples of eight.

4.The input and output data types for the matrices must be either half precision or single precision. (Only HIP_R_16F is shown above, 
but HIP_R_32F also is supported.)

*/

// First, create a cuBLAS handle:
hipblasStatus_t cublasStat = hipblasCreate(&handle);

// Set the math mode to allow cuBLAS to use Tensor Cores:
cublasStat = hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

// Allocate and initialize your matrices (only the A matrix is shown):
size_t matrixSizeA = (size_t)rowsA * colsA;
T_ELEM_IN **devPtrA = 0;

hipMalloc((void**)&devPtrA[0], matrixSizeA * sizeof(devPtrA[0][0]));
T_ELEM_IN A  = (T_ELEM_IN *)malloc(matrixSizeA * sizeof(A[0]));

memset( A, 0xFF, matrixSizeA* sizeof(A[0]));
status1 = hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, rowsA, devPtrA[i], rowsA);

// ... allocate and initialize B and C matrices (not shown) ...

// Invoke the GEMM, ensuring k, lda, ldb, and ldc are all multiples of 8, 
// and m is a multiple of 4:
cublasStat = hipblasGemmEx(handle, transa, transb, m, n, k, alpha,
                          A, HIP_R_16F, lda,
                          B, HIP_R_16F, ldb,
                          beta, C, HIP_R_16F, ldc, HIP_R_32F, algo);
